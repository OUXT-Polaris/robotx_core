#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <NvInfer.h>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <ros/ros.h>
#include <cnn_utils.h>

/*
   TODO 4決め打ちをなんとかする
   TODO 推論制度の改善
   */

using namespace nvinfer1;

class Logger : public ILogger {
  void log(Severity severity, const char * msg) override {
    if (severity != Severity::kINFO)
      ROS_INFO("[[infer.cu]] %s", msg);
  }
} gLogger;

// runtimes
IRuntime *runtime;
ICudaEngine *engine;
IExecutionContext *context;
int inputBindingIndex, outputBindingIndex;
int inputHeight, inputWidth;
Dims inputDims, outputDims;
bool is_initialized = false;
void *bindings[2];

// flags
bool use_mappedMemory;

// pointers
size_t numInput, numOutput;
float *inputDataHost, *outputDataHost;
float *inputDataDevice, *outputDataDevice;

void setup(std::string planFilename, std::string inputName, std::string outputName, bool _use_mappedMemory) {
  ROS_INFO("setup");
  std::ifstream planFile(planFilename.c_str());
  if(!planFile.is_open()) {
    ROS_INFO("cannot get plan file");
    is_initialized = false;
  } else {
    std::stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    std::string plan = planBuffer.str();

    use_mappedMemory = _use_mappedMemory;

    runtime = createInferRuntime(gLogger);
    engine  = runtime->deserializeCudaEngine((void*)plan.data(), plan.size(), nullptr);
    context = engine->createExecutionContext();
    ROS_INFO("load setup finished");

    inputBindingIndex = engine->getBindingIndex(inputName.c_str());
    outputBindingIndex = engine->getBindingIndex(outputName.c_str());
    inputDims = engine->getBindingDimensions(inputBindingIndex);
    outputDims = engine->getBindingDimensions(outputBindingIndex);
    inputHeight = inputDims.d[1];
    inputWidth = inputDims.d[2];
    ROS_INFO("input: h=%d, w=%d", inputHeight, inputWidth);

    numInput = numTensorElements(inputDims);
    numOutput = numTensorElements(outputDims);

    if (use_mappedMemory) {
      // host
      hipHostAlloc(&inputDataHost, numInput * sizeof(float), hipHostMallocMapped);
      hipHostAlloc(&outputDataHost, numOutput * sizeof(float), hipHostMallocMapped);
      // device
      hipHostGetDevicePointer(&inputDataDevice, inputDataHost, 0);
      hipHostGetDevicePointer(&outputDataDevice, outputDataHost, 0);
    } else {
      // host
      inputDataHost = (float*) malloc(numInput * sizeof(float));
      outputDataHost = (float*) malloc(numOutput * sizeof(float));
      // device
      hipMalloc(&inputDataDevice, numInput * sizeof(float));
      hipMalloc(&outputDataDevice, numOutput * sizeof(float));
    }
    bindings[inputBindingIndex] = (void*)inputDataDevice;
    bindings[outputBindingIndex] = (void*)outputDataDevice;

    is_initialized = true;
    ROS_INFO("initialize finished %d, %d", numInput, numOutput);
  }
}

void destroy(void) {
  if(is_initialized) {
    runtime->destroy();
    engine->destroy();
    context->destroy();
    if(use_mappedMemory) {
      hipHostFree(inputDataHost);
      hipHostFree(outputDataHost);
    } else {
      free(inputDataHost);
      free(outputDataHost);
    }
    hipFree(inputDataDevice);
    hipFree(outputDataDevice);
  }
  is_initialized = false;
}

void infer(cv::Mat image, float* out) {
  // cvの画像からcnnを走らせる
  ROS_INFO("get");

  // preprocessing
  cv::resize(image, image, cv::Size(inputWidth, inputHeight));
  cvImageToTensor(image, inputDataHost, inputDims);
  preprocessVgg(inputDataHost, inputDims);

  // execute on cuda
  if (use_mappedMemory) {
    context->execute(1, bindings);
  } else {
    hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
    context->execute(1, bindings);
    hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);
  }

  // output
  /* ROS_INFO("%f %f %f %f", outputDataHost[0], outputDataHost[1], outputDataHost[2], outputDataHost[3]); */
  for (int i = 0; i < 4; i++) {
    out[i] = outputDataHost[i];
  }
}

void test(void) {
  ROS_INFO("inside cu");
  hipDeviceSynchronize();
}

