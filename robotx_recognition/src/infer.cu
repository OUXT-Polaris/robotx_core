#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <ros/ros.h>

// utils
#include <cnn_utils.h>

// TensorRT
#include <NvInfer.h>
#include <NvUffParser.h>

using namespace nvinfer1;
using namespace nvuffparser;

class Logger : public ILogger {
  void log(Severity severity, const char * msg) override {
    if (severity != Severity::kINFO)
      ROS_INFO("[[infer.cu]] %s", msg);
  }
} gLogger;

// runtimes
IRuntime *runtime;
ICudaEngine *engine;
IExecutionContext *context;
int inputBindingIndex, outputBindingIndex;
int inputHeight, inputWidth;
Dims inputDims, outputDims;
bool is_initialized = false;
void *bindings[2];

// flags
bool use_mappedMemory;

// pointers
size_t numInput, numOutput;
float *inputDataHost, *outputDataHost;
float *inputDataDevice, *outputDataDevice;

int convert(std::string uffFilename, std::string planFilename, std::string inputName, std::string outputName) {
  IBuilder *builder = createInferBuilder(gLogger);
  INetworkDefinition *network = builder->createNetwork();
  IUffParser *parser = createUffParser();

  parser->registerInput(inputName.c_str(), DimsCHW(3, 224, 224));
  parser->registerOutput(outputName.c_str());
  parser->parse(uffFilename.c_str(), *network, DataType::kFLOAT);  // or, kHALF

  builder->setMaxBatchSize(1);
  builder->setMaxWorkspaceSize(1<<20);
  ICudaEngine *_engine = builder->buildCudaEngine(*network);

  ofstream f;
  f.open(planFilename.c_str());  // plan
  IHostMemory *serializedEngine = _engine->serialize();
  f.write((char *)serializedEngine->data(), serializedEngine->size());
  f.close();

  builder->destroy();
  parser->destroy();
  network->destroy();
  _engine->destroy();
  serializedEngine->destroy();

  return 0;
}

int setup(std::string planFilename, std::string inputName, std::string outputName, bool _use_mappedMemory) {
  ROS_INFO("setup");
  std::ifstream planFile(planFilename.c_str());
  if(!planFile.is_open()) {
    ROS_INFO("cannot get plan file");
    is_initialized = false;
    return -1;
  } else {
    std::stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    std::string plan = planBuffer.str();

    use_mappedMemory = _use_mappedMemory;

    runtime = createInferRuntime(gLogger);
    engine  = runtime->deserializeCudaEngine((void*)plan.data(), plan.size(), nullptr);
    context = engine->createExecutionContext();
    ROS_INFO("load setup finished");

    inputBindingIndex = engine->getBindingIndex(inputName.c_str());
    outputBindingIndex = engine->getBindingIndex(outputName.c_str());
    inputDims = engine->getBindingDimensions(inputBindingIndex);
    outputDims = engine->getBindingDimensions(outputBindingIndex);
    inputHeight = inputDims.d[1];
    inputWidth = inputDims.d[2];
    ROS_INFO("input: h=%d, w=%d", inputHeight, inputWidth);

    numInput = numTensorElements(inputDims);
    numOutput = numTensorElements(outputDims);

    if (use_mappedMemory) {
      // host
      hipHostAlloc(&inputDataHost, numInput * sizeof(float), hipHostMallocMapped);
      hipHostAlloc(&outputDataHost, numOutput * sizeof(float), hipHostMallocMapped);
      // device
      hipHostGetDevicePointer(&inputDataDevice, inputDataHost, 0);
      hipHostGetDevicePointer(&outputDataDevice, outputDataHost, 0);
    } else {
      // host
      inputDataHost = (float*) malloc(numInput * sizeof(float));
      outputDataHost = (float*) malloc(numOutput * sizeof(float));
      // device
      hipMalloc(&inputDataDevice, numInput * sizeof(float));
      hipMalloc(&outputDataDevice, numOutput * sizeof(float));
    }
    bindings[inputBindingIndex] = (void*)inputDataDevice;
    bindings[outputBindingIndex] = (void*)outputDataDevice;

    is_initialized = true;
    ROS_INFO("initialize finished %d, %d", numInput, numOutput);

    return (int)numOutput;
  }
}

void destroy(void) {
  if(is_initialized) {
    runtime->destroy();
    engine->destroy();
    context->destroy();
    if(use_mappedMemory) {
      hipHostFree(inputDataHost);
      hipHostFree(outputDataHost);
    } else {
      free(inputDataHost);
      free(outputDataHost);
    }
    hipFree(inputDataDevice);
    hipFree(outputDataDevice);
  }
  is_initialized = false;
}

void infer(cv::Mat image, float* out) {
  // cvの画像からcnnを走らせる
  // preprocessing
  cv::resize(image, image, cv::Size(inputWidth, inputHeight));
  cvImageToTensor(image, inputDataHost, inputDims);
  preprocessVgg(inputDataHost, inputDims);

  // execute on cuda
  if (use_mappedMemory) {
    context->execute(1, bindings);
  } else {
    hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
    context->execute(1, bindings);
    hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);
  }

  // output
  /* ROS_INFO("%f %f %f %f", outputDataHost[0], outputDataHost[1], outputDataHost[2], outputDataHost[3]); */
  for (int i = 0; i < (int)numOutput; i++) {
    out[i] = outputDataHost[i];
  }
}

void test(void) {
  ROS_INFO("inside cu");
  hipDeviceSynchronize();
}

